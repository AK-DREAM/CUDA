
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
using namespace std;

const int N = 10000005;
int a[N];

__global__ void sort_kernel(int *d_vec, int n, int tp) {
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    i = 2*i+tp;
    if (i+1 < n) {
        if (d_vec[i] > d_vec[i+1]) {
            int tmp = d_vec[i];
            d_vec[i] = d_vec[i+1];
            d_vec[i+1] = tmp;
        }
    }
}

void sort(int *a, int n) {
    int* d_vec;
    hipMalloc(&d_vec, n*sizeof(int));
    hipMemcpy(d_vec, a, n*sizeof(int), hipMemcpyHostToDevice);

    for (int i = 0; i < n; i++) {
        sort_kernel<<<(n+127)/128,128>>>(d_vec, n, i&1);
        hipDeviceSynchronize();
    }

    hipMemcpy(a, d_vec, n*sizeof(int), hipMemcpyDeviceToHost);
}

int main() {
    int n; //scanf("%d", &n);
    n = 10000000;
    for (int i = 0; i < n; i++) {
        a[i] = rand();
        // scanf("%d", &a[i]);
    }
    double t = 1.0*clock()/CLOCKS_PER_SEC;
    sort(a, n);
    printf("%lf\n", 1.0*clock()/CLOCKS_PER_SEC-t);
    // for (int i = 0; i < n; i++) printf("%d ", a[i]);
    return 0;
}