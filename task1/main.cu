
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
using namespace std;

#define PERF_GPU(msg, behavior)             \
    {                                       \
        hipEvent_t start1;                 \
        hipEventCreate(&start1);           \
        hipEvent_t stop1;                  \
        hipEventCreate(&stop1);            \
        hipEventRecord(start1, NULL);      \
        behavior                            \
        hipEventRecord(stop1, NULL);       \
        hipEventSynchronize(stop1);        \
        float msecTotal1 = 0.0f;            \
        hipEventElapsedTime(&msecTotal1, start1, stop1);   \
        printf("GPU time: %f\n", msecTotal1);  \
    } 

__global__ void reduce_sum_kernel(float *d_in, int n, int dim, int tot) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < tot) {
        d_in[idx] += d_in[idx+tot];  
    }
}

void reduce_sum(float *h_in, float *h_out, size_t n, size_t dim) {
    float* d_in;
    hipMalloc(&d_in, n*dim*sizeof(float));
    hipMemcpy(d_in, h_in, n*dim*sizeof(float), hipMemcpyHostToDevice);

    auto work = [&]{
        for (int i = n; i >= 2; i >>= 1) {
            int tot = i*dim/2;
            reduce_sum_kernel<<<(tot+1023)/1024,1024>>>(d_in, i, dim, tot);
            hipDeviceSynchronize();
        }
    };
    PERF_GPU(
        "gpu impl",
        work();
    );
    
    hipMemcpy(h_out, d_in, dim*sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_in); 
}

const int N = 100000005;
float a[N], b[N];

int main() {
    int n, dim; 
    // scanf("%d %d", &n, &dim);
    n = 1000, dim = 100;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < dim; j++) {
            a[i*dim+j] = i*dim+j;
            // scanf("%f", &a[i*dim+j]);
        }
    }
    int lim = 1;
    while (lim < n) lim <<= 1;
    
    reduce_sum(a, b, lim, dim);
    for (int i = 0; i < dim; i++) printf("%f ", b[i]);
    puts("");
    return 0;
}